
#include <hip/hip_runtime.h>
#include <iostream>
#define N 10000000
__global__ void vector_add(float* out, float* a, float* b, int n) {
  int stride = 1;
  for (int i = 0; i < n; i += stride) {
    out[i] = a[i] + b[i];
  }
}

__global__ void cuda_hello() {
  printf("Hello from block %d, thread %d\n", blockDim.x, threadIdx.x);
}
int main() {
  cuda_hello<<<1, 4>>>();
  float *a, *b, *out;
  float *d_a, *d_b, *d_out;
  a = (float*)malloc(N * sizeof(float));
  b = (float*)malloc(N * sizeof(float));
  for (int i = 0; i < N; i++) {
    a[i] = 2;
  }
  for (int i = 0; i < N; i++) {
    b[i] = 4;
  }
  out = (float*)malloc(N * sizeof(float));
  hipMalloc((void**)&d_b, sizeof(float) * N);
  hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_out, sizeof(float) * N);
  hipMemcpy(d_out, out, sizeof(float) * N, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_a, sizeof(float) * N);
  hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
  vector_add<<<1, 1>>>(d_out, d_a, d_b, N);
  hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
  printf("out[0] = %f\n", out[0]);
  hipFree(d_a);
  free(a);
  return 0;
}