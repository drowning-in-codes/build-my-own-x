#include "hip/hip_runtime.h"
#include <iostream>
#define N 100000
__global__ void addOfblocks(int n, float *x, float *y) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = idx; i < n; i += stride) y[i] = x[i] + y[i];
}
__global__ void addOfThreads(int n, float *x, float *y) {
  int idx = threadIdx.x;
  int stride = blockDim.x;
  for (int i = idx; i < n; i += blockDim) y[i] = x[i] + y[i];
}
int main() {
  float *a, *out;

  hipMallocManaged(&a, N * sizeof(float));
  hipMallocManaged(&out, N * sizeof(float));
  for (int i = 0; i < N; i++) {
    a[i] = 2;
    b[i] = 4;
  }

  add<<<2, 4>>>(N, a, b);
  hipDeviceSynchronize();

  hipFree(a);
  hipFree(b);
  return 0;
}